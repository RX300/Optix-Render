#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void init_rng(hiprandState* state, unsigned long long seed)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &state[id]);
}

extern "C" void initOneState(hiprandState* devStates) {
	init_rng << <1, 1 >> > (devStates, 1);
}